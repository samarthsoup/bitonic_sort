#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <algorithm>
#include <iostream>
#include <fstream>

#define MAX_THREADS_PER_BLOCK 1024

const char* default_input_filename = "data/generated_data.txt";
const char* default_output_filename = "data/output.txt";

void bitonicSortCPU(int* arr, int n) 
{
    for (int k = 2; k <= n; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            for (int i = 0; i < n; ++i) {
                int ij = i ^ j;

                if (ij > i) {
                    if ((i & k) == 0) {
                        if (arr[i] > arr[ij]){
                            std::swap(arr[i], arr[ij]);
                        }
                    } else {
                        if (arr[i] < arr[ij]){
                            std::swap(arr[i], arr[ij]);
                        }
                    }
                }
            }
        }
    }
}

__global__ void bitonicSortGPU(int* arr, int j, int k)
{
    unsigned int i, ij;

    i = threadIdx.x + blockDim.x * blockIdx.x;

    ij = i ^ j;

    if (ij > i) {
        if ((i & k) == 0) {
            if (arr[i] > arr[ij]) {
                std::swap(arr[i], arr[ij]);
            }
        } else {
            if (arr[i] < arr[ij]) {
                std::swap(arr[i], arr[ij]);
            }
        }
    }
}

void printArray(int* arr, int size) 
{
    for (int i = 0; i < size; ++i)
        std::cout << arr[i] << " ";
    std::cout << std::endl;
}

bool isSorted(int* arr, int size) 
{
    for (int i = 1; i < size; ++i) 
    {
        if (arr[i] < arr[i - 1])
            return false;
    }
    return true;
}

bool isPowerOfTwo(int num) 
{
    return num > 0 && (num & (num - 1)) == 0;
}

int nextPowerOfTwo(int n) 
{
    if (n && !(n & (n - 1))) {
        return n;
    }
    
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;

    return n;
}

int main(int argc, char* argv[]) 
{   
    const char* input_filename = default_input_filename;
    const char* output_filename = default_output_filename;

    for (int i = 1; i < argc; ++i) {
        if (std::strcmp(argv[i], "--input") == 0 && i + 1 < argc) {
            input_filename = argv[++i];
        } else if (std::strcmp(argv[i], "--output") == 0 && i + 1 < argc) {
            output_filename = argv[++i];
        } else {
            std::cerr << "usage: " << argv[0] << " [--input filename] [--output filename]\n";
            return 1;
        }
    }

    std::ifstream infile(input_filename);
    if (!infile) {
        std::cerr << "error opening file: " << input_filename << "\n";
        return 1;
    }

    int input_size = 0;
    infile >> input_size;

    int size;
    if (input_size <= 0) {
        std::cerr << "array size must be a positive integer\n";
        return 1;
    }

    if (!isPowerOfTwo(input_size)) {   
        std::cout << "size provided is not a power of two, size will be the next power of two and remaining spots of the array will be padded with zeroes\nSize provided: " << input_size << std::endl;
        size = nextPowerOfTwo(input_size);
        std::cout << "the nearest higher power of two is: " << size << std::endl;
    } else {
        size = input_size;
    }

    int* arr = new int[size];
    int* carr = new int[size];
    int* temp = new int[size];

    int* gpuArrbiton;
    int* gpuTemp;

    srand(static_cast<unsigned int>(time(nullptr)));
    for (int i = 0; i < input_size; ++i) {
        if (!(infile >> arr[i])) {
            std::cerr << "error reading number at position " << i + 1 << ".\n";
            delete[] arr;
            return 1;
        }
        carr[i] = arr[i];
    }

    infile.close();

    for (int i = input_size; i < size; ++i) {
        arr[i] = 0;
        carr[i] = 0;
    }

    hipMalloc((void**)&gpuTemp, size * sizeof(int));
    hipMalloc((void**)&gpuArrbiton, size * sizeof(int));

    hipMemcpy(gpuArrbiton, arr, size * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    float GPU_time_ms = 0;

    clock_t startCPU, endCPU;

    int threadsPerBlock = MAX_THREADS_PER_BLOCK;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int j, k;

    hipEventRecord(startGPU);
    for (k = 2; k <= size; k <<= 1) {
        for (j = k >> 1; j > 0; j = j >> 1) {
            bitonicSortGPU <<<blocksPerGrid, threadsPerBlock>>> (gpuArrbiton, j, k);
        }
    }
    hipEventRecord(stopGPU);

    hipMemcpy(arr, gpuArrbiton, size * sizeof(int), hipMemcpyDeviceToHost);
    hipEventSynchronize(stopGPU);
    hipEventElapsedTime(&GPU_time_ms, startGPU, stopGPU);

    startCPU = clock();
    bitonicSortCPU(carr, size);
    endCPU = clock();

    double CPU_time_ms = static_cast<double>(endCPU - startCPU) / (CLOCKS_PER_SEC / 1000.0);
    
    if (isSorted(arr, size))
        std::cout << "\n\nsort checker: gpu array success" << std::endl;
    else
        std::cout << "sort checker: gpu array fail" << std::endl;
   
    if (isSorted(carr, size))
        std::cout << "sort checker: cpu array success" << std::endl;
    else
        std::cout << "sort checker: cpu array fail" << std::endl;

    std::cout << "\n\ngpu time: " << GPU_time_ms << " ms" << std::endl;
    std::cout << "cpu time: " << CPU_time_ms << " ms" << std::endl;

    std::ofstream outfile(output_filename);
    if (!outfile) {
        std::cerr << "error opening output file.\n";
        delete[] arr;
        return 1;
    }

    for (int i = size-input_size; i < size; ++i) {
        outfile << arr[i] << " ";
    }

    outfile.close();

    delete[] arr;
    delete[] carr;
    delete[] temp;

    hipFree(gpuArrbiton);
    hipFree(gpuTemp);

    return 0;
}
